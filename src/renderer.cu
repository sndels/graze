#include "hip/hip_runtime.h"
#include "renderer.hu"

#include "cuda_helpers.hu"

namespace {
    __global__ void cuRender(Film::Surface surface)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= surface.width || y >= surface.height)
            return;
        const int pxI = (y * surface.width + x) * 3;

        surface.fb[pxI] = float(x) / surface.width;
        surface.fb[pxI + 1] = float(y) / surface.height;
        surface.fb[pxI + 2] = 0.2;
    }
}

void render(Film* film)
{
    const auto& surface = film->surface();
    const uint32_t tx = 8;
    const uint32_t ty = 8;
    const dim3 blocks{
        surface.width / tx + 1,
        surface.height / ty + 1
    };
    const dim3 threads{tx, ty};
    // This passes a copy of surface but the contained pointer won't be modified, only data
    cuRender<<<blocks, threads>>>(surface);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    film->setDirty();
}
