#include "hip/hip_runtime.h"
#include "stdio.h"

#include "cuda_helpers.hu"
#include "film.hu"
#include "gui.hu"
#include "material.hu"
#include "renderer.hu"
#include "sphere.hu"
#include "timer.hpp"
#include "window.hpp"

namespace {
    // This a loose upper bound
    const uint32_t numSpheres = 500;

    // Init on gpu to use abstract base class
    __global__ void init_scene(Material** materials, Intersectable** intersectables, Intersectable** scene)
    {
        materials[0] = new Lambertian{Vec3{0.5f, 0.5f, 0.5f}};
        intersectables[0] = new Sphere{
            Vec3{0.f, -1000.f, 0.f},
            1000.f,
            materials[0]
        };

        int i = 1;
        hiprandState randState;
        hiprand_init(1337, 0, 0, &randState);
        for (int a = -11; a < 11; ++a) {
            for (int b = -11; b < 11; ++b) {
                const Vec3 center{
                    a + 0.9f * hiprand_uniform(&randState),
                    0.2f,
                    b + 0.9f * hiprand_uniform(&randState)
                };
                if (len(center - Vec3{4.f, 0.2f, 0.f}) > 0.9f) {
                    const float chooseMat = hiprand_uniform(&randState);
                    if (chooseMat < 0.8f) {
                        materials[i] = new Lambertian{
                            Vec3{
                                hiprand_uniform(&randState) * hiprand_uniform(&randState),
                                hiprand_uniform(&randState) * hiprand_uniform(&randState),
                                hiprand_uniform(&randState) * hiprand_uniform(&randState)
                            }
                        };
                    } else if (chooseMat < 0.95f) {
                        materials[i] = new Metal{
                            0.5f * (1.f - Vec3{
                                hiprand_uniform(&randState),
                                hiprand_uniform(&randState),
                                hiprand_uniform(&randState)
                            }),
                            0.5f * hiprand_uniform(&randState)
                        };
                    } else
                        materials[i] = new Dielectric{1.5f};

                    intersectables[i] = new Sphere{
                        center,
                        0.2f,
                        materials[i]
                    };
                    ++i;
                }
            }
        }

        materials[i] = new Dielectric{1.5f};
        intersectables[i] = new Sphere{
            Vec3{0.f, 1.f, 0.f},
            1.f,
            materials[i]
        };
        materials[++i] = new Lambertian{Vec3{0.4f, 0.2f, 0.1f}};
        intersectables[i] = new Sphere{
            Vec3{-4.f, 1.f, 0.f},
            1.f,
            materials[i]
        };
        materials[++i] = new Metal{Vec3{0.7f, 0.6f, 0.5f}, 0.f};
        intersectables[i] = new Sphere{
            Vec3{4.f, 1.f, 0.f},
            1.f,
            materials[i]
        };

        *scene = new IntersectableList(intersectables, ++i);
    }

    __global__ void free_scene(Material** materials, Intersectable** intersectables, Intersectable** scene)
    {
        for (int i = 0; i < reinterpret_cast<IntersectableList*>(*scene)->numIntersectables; ++i) {
            delete materials[i];
            delete intersectables[i];
        }
        delete *scene;
    }
}

int main()
{
    Window window{1280, 720};
    if (!window.init())
        return -1;

    GUI gui{window.ptr()};
    Film film{gui.filmSettings()};
    Timer timer;

    timer.reset();
    Material** materials;
    Intersectable** intersectables;
    Intersectable** scene;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&materials), (numSpheres + 1) * sizeof(Material*)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&intersectables), (numSpheres + 1) * sizeof(Intersectable*)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&scene), sizeof(Intersectable*)));
    init_scene<<<1, 1>>>(materials, intersectables, scene);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    printf("Scene built in %.3fs!\n", timer.seconds());


    // Run the main loop
    while (window.open()) {
        window.startFrame();
        gui.startFrame();

        // Prepare GL
        glClear(GL_COLOR_BUFFER_BIT);

        if (window.startRender() || gui.startRender()) {
            film.updateSettings(gui.filmSettings());

            printf("Initiating render!\n");
            timer.reset();
            render(gui.cameraSettings(), &film, scene);
            printf("Done in %.3fs!\n", timer.seconds());
        }

        film.display(window.width(), window.height());
        gui.endFrame();
        window.endFrame();
    }

    checkCudaErrors(hipDeviceSynchronize());
    free_scene<<<1, 1>>>(materials, intersectables, scene);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(materials));
    checkCudaErrors(hipFree(intersectables));
    checkCudaErrors(hipFree(scene));
    film.destroy();
    gui.destroy();
    window.destroy();
    return 0;
}
