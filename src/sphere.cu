#include "hip/hip_runtime.h"
#include "sphere.hu"

__device__ Sphere::Sphere(const Vec3& center, const float radius, Material* material) :
    _center0{center},
    _center1{center},
    _time0{0.f},
    _time1{1.f},
    _radius{radius},
    _material{material}
{ }

__device__ Sphere::Sphere(const Vec3& center0, const Vec3& center1, const float time0, const float time1, const float radius, Material* material) :
    _center0{center0},
    _center1{center1},
    _time0{time0},
    _time1{time1},
    _radius{radius},
    _material{material}
{ }

__device__ bool Sphere::intersect(Ray* r, Hit* hit) const
{
    const Vec3 center = lerp(_center0, _center1, (r->time - _time0) / (_time1 - _time0));
    const Vec3 oc = r->o - center;
    const float a = lenSq(r->d);
    const float b = 2.f * dot(oc, r->d);
    const float c = lenSq(oc) - _radius * _radius;
    const float d = b * b - 4.f * a * c;
    // Check for hit
    if (d > 0.f) {
        // Get the two hit locations, t0 < t1
        float t0 = (-b - sqrt(d)) / (2.f * a);
        float t1 = (-b + sqrt(d)) / (2.f * a);
        if (t0 > t1) {
            float tmp = t0;
            t0 = t1;
            t1 = tmp;
        }

        // Check if sphere is outside ray range
        if (t0 > r->tMax || t1 <= r->tMin)
            return false;

        float t = t0;
        // Check if we are inside
        if (t <= r->tMin) {
            t = t1;
            // Check if outgoing hit is beyond ray range
            if (t > r->tMax)
                return false;
        }
        r->tMax = t;
        hit->t = t;
        hit->p = r->point(t);
        hit->n = (hit->p - center) / _radius;
        hit->material = _material;
        return true;
    }
    return false;
}
